
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <cudnn.h>
#include <chrono>

// CUDA runtime

#include <hipblas.h>

#include <sstream>
#include <iostream>

using namespace std;

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != CUDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << cudnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors2(val) check((val), #val, __FILE__, __LINE__)


hipDeviceProp_t prop;
int devid = -1;

extern "C" __declspec(dllexport) void NSetDevice(int gpuid)
{
	if (devid != gpuid)
	{
		devid = gpuid;
		checkCudaErrors(hipGetDeviceProperties(&prop, devid));
		checkCudaErrors(hipSetDevice(gpuid));
	}
}

extern "C" __declspec(dllexport) int NGetDevice()
{
	return devid;
}

extern "C" __declspec(dllexport) void* NAllocate(int bytesize, int gpuid)
{
	NSetDevice(gpuid);
	void* a = 0;
	checkCudaErrors(hipMalloc((void**)&a, bytesize));
	if (a == 0)
		FatalError("Allocation Error on GPU device!");
	return a;
}

extern "C" __declspec(dllexport) void NFree(void* arr)
{
	checkCudaErrors(hipFree(arr));
}

extern "C" __declspec(dllexport) void NCopyFromHostToGPU(void* src, void* dst, int bytesize)
{
	checkCudaErrors(hipMemcpy(dst, src, bytesize, hipMemcpyHostToDevice));
}

extern "C" __declspec(dllexport) void NCopyFromGPUToHost(void* src, void* dst, int bytesize)
{
	checkCudaErrors(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToHost));
}

extern "C" __declspec(dllexport) void NCheckError()
{
	checkCudaErrors(hipGetLastError());
}

extern "C" __declspec(dllexport) void NDeviceSynchronize()
{
	checkCudaErrors(hipDeviceSynchronize());
}

const int arraySize = 2000000;
const float a[arraySize] = { 0.1f, 0.2f, 0.3f, 0.4f, 0.1f };
const float b[arraySize] = { 0, 0, 0, 0, 0 };
float c[arraySize] = { 1,2,3,4,5 };


void addWithCuda(float* c, const float* a, const float* b, unsigned int size);
void addWithCublas(hipblasHandle_t* h, float* c, const float* a, const float* b, unsigned int size);

int main()
{
	float* dev_a = (float*)NAllocate(arraySize * sizeof(float), 0);
	float* dev_b = (float*)NAllocate(arraySize * sizeof(float), 0);
	float* dev_c = (float*)NAllocate(arraySize * sizeof(float), 0);

	NCopyFromHostToGPU((void*)a, dev_a, arraySize * sizeof(float));
	NCopyFromHostToGPU((void*)b, dev_b, arraySize * sizeof(float));

	hipblasHandle_t handle;
	checkCudaErrors(hipblasCreate(&handle));

	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
	for (int i = 0; i < 25000; i++)
	{
		addWithCuda(dev_c, dev_a, dev_c, arraySize);
	}
	NDeviceSynchronize();

	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	NCheckError();

	std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "ms" << std::endl;
	
	NCopyFromGPUToHost(dev_c, c, arraySize * sizeof(float));
	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
		c[0], c[1], c[2], c[3], c[4]);

	std::chrono::steady_clock::time_point begin2 = std::chrono::steady_clock::now();
	for (int i = 0; i < 25000; i++)
	{
		addWithCublas(&handle, dev_c, dev_a, dev_c, arraySize);
	}
	NDeviceSynchronize();

	std::chrono::steady_clock::time_point end2 = std::chrono::steady_clock::now();

	NCheckError();
	checkCudaErrors(hipblasDestroy(handle));

	std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::milliseconds>(end2 - begin2).count() << "ms" << std::endl;


	NCopyFromGPUToHost(dev_c, c, arraySize * sizeof(float));
	NFree(dev_a);
	NFree(dev_b);
	NFree(dev_c);

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
		c[0], c[1], c[2], c[3], c[4]);

	auto cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}


__global__ void addSingleArrays2Kernel(float* c, const float* a, const float* b, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
		c[i] = a[i] + b[i];
}

inline void addWithCuda(float* c, const float* a, const float* b, unsigned int size)
{
	int th = prop.maxThreadsPerBlock;

	if (size < th)
		th = size;

	addSingleArrays2Kernel <<<(size + th - 1) / th, th >>> (c, a, b, size);
}


inline void addWithCublas(hipblasHandle_t* h ,float* c, const float* a, const float* b, unsigned int size)
{
	float alpha = 1;
	hipblasSaxpy(*h, arraySize, &alpha, a, 1, c, 1);
	//cublasSaxpy(*h, arraySize, &alpha, b, 1, c, 1);
}
